﻿#include <iostream>
#include <string>
#include <iomanip>
#include "hip/hip_runtime.h"


// #define imin(a, b) (a<b?a:b)

//定义X,Y,Z各维的长度
const int dimX = 10;
const int dimY = 10;
const int dimZ = 10;
const int SIZE = dimX * dimY * dimZ;

//设置每个线程块中线程数量，此处设置三维一样
const int threadPerBlock = 32;

//设置迭代次数
const int times = 10;

//设置stencil边界处邻居的值
__device__ const double BORDER = 0.0;

int count = 0;


//设定线程格中线程块的数量, 避免启动过多线程块
int blockPerGrid(const int dim, const int threadPerBlock)
{
   //由于暂时一个线程只计算一个stencil，所以暂时不能指定线程块的限制
   int temp = dim / threadPerBlock;
   if (dim % threadPerBlock != 0) {
      temp += 1; 
   }
   return temp;
}

//错误处理
#define CHECK_ERROR(error) checkCudaError(error, __FILE__, __LINE__)
#define CHECK_STATE(msg) checkCudaState(msg, __FILE__, __LINE__)

inline void checkCudaError(hipError_t error, const char *file, const int line)
{
   if (error != hipSuccess) {
      std::cerr << "CUDA CALL FAILED:" << file << "( " << line << ")- " << hipGetErrorString(error) << std::endl;
      exit(EXIT_FAILURE);
   }
}
inline void checkCudaState(const char *msg, const char *file, const int line)
{
   hipError_t error = hipGetLastError();
   if (error != hipSuccess) {
      std::cerr << "---" << msg << " Error---" << std::endl;
      std::cerr << file << "( " << line << ")- " << hipGetErrorString(error) << std::endl;
      exit(EXIT_FAILURE);
   }
}


//计算线程与元素的唯一偏移，以x为行，y为列，z为高
__device__ __host__ int offset(int x, int y, int z) 
{
   return (((x + dimX) % dimX) + ((y + dimY) % dimY) * dimX + ((z + dimZ) % dimZ) * dimX * dimY);
}

__global__ void kernel(double *dev_grid_in, double *dev_grid_out)
{
   //使用shared memory存储每个线程块中的计算
   __shared__ double cache[threadPerBlock * threadPerBlock * 1];
   int cacheIndex = threadIdx.x * threadIdx.y * threadIdx.z;
   cache[cacheIndex] = 0.0;
   __syncthreads();

   //线程索引
   int x = threadIdx.x + blockIdx.x * blockDim.x;
   int y = threadIdx.y + blockIdx.y * blockDim.y;
   int z = threadIdx.z + blockIdx.z * blockDim.z;

   //设置stencil中各元素值
   double center = dev_grid_in[offset(x, y, z)];
   double up     = (z < (dimZ - 1)) ? dev_grid_in[offset(x, y, z + 1)] : BORDER;
   double down   = (z > 0) ? dev_grid_in[offset(x, y, z - 1)] : BORDER;
   double west   = (x > 0) ? dev_grid_in[offset(x - 1, y, z)] : BORDER;
   double east   = (x < (dimX - 1)) ? dev_grid_in[offset(x + 1, y, z)] : BORDER;
   double south  = (y > 0) ? dev_grid_in[offset(x, y - 1, z)] : BORDER;
   double north  = (y < (dimY - 1)) ? dev_grid_in[offset(x, y + 1, z)] : BORDER;

   //    dev_grid_out[offset(x, y, z)] = 1.0;
//   dev_grid_out[offset(x, y, z)] = (center + up + down + west + east + south + north) * (1.0 / 7.0);
   cache[cacheIndex] = (center + up + down + west + east + south + north) * (1.0 / 7.0);
   __syncthreads();

   //显然此处没有加速效果
   dev_grid_out[offset(x, y, z)] = cache[cacheIndex];
}

//初始化输入，输出
void init(double *grid, int dimX, int dimY, int dimZ)
{
   for (int z=0; z<dimZ; ++z) {
      for (int y=0; y<dimY; ++y) {
         for (int x=0; x<dimX; ++x) {
            if ((x*y*z == 0) || (x == dimX-1) || (y == dimY-1) || (z == dimZ-1)) {
               grid[offset(x, y, z)] = 7;
            }
            else {
               grid[offset(x, y, z)] = 0;
               //      grid[offset(x, y, z)] = count;
            }
            count++;
         }
      }
   }
}

void print(double *grid)
{
   for (int z=0; z<dimZ; ++z) {
      std::cout << z << ":\n\n";
      for (int y=0; y<dimY; ++y) {
         for (int x=0; x<dimX; ++x) {
            std::cout << std::fixed << std::setprecision(3) << grid[offset(x, y, z)] << "\t";
         }
         std::cout << std::endl;
      }
      std::cout << std::endl;
   }
}

void debug(int test, std::string str)
{

   if (test != 0) {
      std::cout << "-----------" << str  << "--------------" << std::endl;
      std::cout << test << std::endl;
   }
   else {
      std::cout << "-----------" << str  << "--------------" << std::endl;
   }
}

int main(void)
{
   CHECK_ERROR(hipSetDevice(0));
   //由于blocks不能大于1024，所以最后一维设备为1
   dim3 blocks(threadPerBlock, threadPerBlock, 1);
   dim3 grids(blockPerGrid(dimX, blocks.x), blockPerGrid(dimY, blocks.y), blockPerGrid(dimZ, blocks.z));

   double *grid_in, *grid_out;
   grid_in = (double *)malloc(SIZE * sizeof(double));
   grid_out = (double *)malloc(SIZE * sizeof(double));

   double *dev_grid_in, *dev_grid_out;
   CHECK_ERROR(hipMalloc((void**)&dev_grid_in, SIZE * sizeof(double)));
   CHECK_ERROR(hipMalloc((void**)&dev_grid_out, SIZE * sizeof(double)));

   init(grid_in, dimX, dimY, dimZ);
   init(grid_out, dimX, dimY, dimZ);

//   debug(0, "input");
//   print(grid_in);

   //统计用于GPU计算的时间
   hipEvent_t start, stop;
   CHECK_ERROR(hipEventCreate(&start));
   CHECK_ERROR(hipEventCreate(&stop));
   CHECK_ERROR(hipEventRecord(start, 0));
   CHECK_ERROR(hipEventSynchronize(start));

   CHECK_ERROR(hipMemcpy(dev_grid_in, grid_in, SIZE * sizeof(double), hipMemcpyHostToDevice));
   CHECK_ERROR(hipMemcpy(dev_grid_out, grid_out, SIZE * sizeof(double), hipMemcpyHostToDevice));

   for (int i=0; i<times; ++i) {
      kernel<<<grids, blocks>>>(dev_grid_in, dev_grid_out);
      std::swap(dev_grid_in, dev_grid_out);
   }
   hipDeviceSynchronize();

   CHECK_STATE("kernel call");

   CHECK_ERROR(hipMemcpy(grid_in, dev_grid_in, SIZE * sizeof(double), hipMemcpyDeviceToHost));

   //计算统计的时间
   CHECK_ERROR(hipEventRecord(stop, 0));
   CHECK_ERROR(hipEventSynchronize(stop));
   float elapsedTime;
   CHECK_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

   debug(0, "output");
   print(grid_in);

   std::cout << "Time elapsed: " << std::fixed << std::setprecision(6) << elapsedTime << " ms"  << std::endl;

   CHECK_ERROR(hipEventDestroy(start));
   CHECK_ERROR(hipEventDestroy(stop));

   free(grid_in);
   free(grid_out);
   CHECK_ERROR(hipFree(dev_grid_in));
   CHECK_ERROR(hipFree(dev_grid_out));
   return 0;
}


